#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <string>
#include <custatevec.h> // custatevecInitializeStateVector
#include <hip/hip_complex.h>
#include <random>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iomanip>

#define INV_SQRT2 (0.7071067811865475) // Approximation of 1/sqrt(2)

// Macro to check CUDA API errors
#define CHECK_CUDA(func)                                                          \
    {                                                                             \
        hipError_t status = (func);                                              \
        if (status != hipSuccess)                                                \
        {                                                                         \
            printf("CUDA API failed at line %d in file %s with error: %s (%d)\n", \
                   __LINE__, __FILE__, hipGetErrorString(status), status);       \
            return EXIT_FAILURE;                                                  \
        }                                                                         \
    }

// Macro to check cuSPARSE API errors
#define CHECK_CUSPARSE(func)                                                     \
    {                                                                            \
        hipsparseStatus_t status = (func);                                        \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                   \
        {                                                                        \
            printf("CUSPARSE API failed at line %d in file %s with error: %d\n", \
                   __LINE__, __FILE__, status);                                  \
            return EXIT_FAILURE;                                                 \
        }                                                                        \
    }

// Macro to check cuSPARSE API errors
#define CHECK_CUSTATEVECTOR(func)                                                     \
    {                                                                                 \
        custatevecStatus_t status = (func);                                           \
        if (status != CUSTATEVEC_STATUS_SUCCESS)                                      \
        {                                                                             \
            printf("CUSTATEVECTOR API failed at line %d in file %s with error: %d\n", \
                   __LINE__, __FILE__, status);                                       \
            return EXIT_FAILURE;                                                      \
        }                                                                             \
    }

#define CHECK_BROAD_ERROR(integer)                                    \
    {                                                                 \
        if (integer != 0)                                             \
        {                                                             \
            printf("Broad CUDA ERROR failed at line %d in file %s\n", \
                   __LINE__, __FILE__);                               \
            return EXIT_FAILURE;                                      \
        }                                                             \
    }

void generateRandomArray(double *arr, std::size_t size)
{
    std::random_device rd;                                   // Seed
    std::mt19937 gen(rd());                                  // Mersenne Twister RNG
    std::uniform_real_distribution<double> dist(0.0f, 1.0f); // Range [0, 1)

    for (std::size_t i = 0; i < size; ++i)
    {
        arr[i] = dist(gen);
    }
}

int main()
{
    const int nIndexBits = 30;
    // const auto cuStateVecComputeType = CUSTATEVEC_COMPUTE_64F;
    // auto cuStateVecCudaDataType = HIP_C_64F;
    // using cuType = hipDoubleComplex;
    using cuType = hipComplex;
    const auto cuStateVecComputeType = CUSTATEVEC_COMPUTE_32F;
    const auto cuStateVecCudaDataType = HIP_C_32F;

    constexpr int svSize = (1 << nIndexBits);

    const int nShots = 100;
    const int nMaxShots = nShots;
    int bitOrdering[nIndexBits] = {};
    for (int i = 0; i < nIndexBits; ++i)
    {
        bitOrdering[i] = i;
    }
    const int bitStringLen = nIndexBits;
    custatevecIndex_t bitStrings[nShots];
    double randnums[nShots] = {};
    generateRandomArray(randnums, nShots);

    cuType xMat[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};
    cuType zMat[] = {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {-1.0, 0.0}};
    cuType hMat[] = {{INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {-INV_SQRT2, 0.0}};

    cuType h_sv[1];
    cuType *d_sv;
    CHECK_CUDA(hipMalloc((void **)&d_sv, svSize * sizeof(cuType)));

    //----------------------------------------------------------------------------------------------

    {
        auto start_m = std::chrono::high_resolution_clock::now();

        int controlsAll[nIndexBits];
        int controlsAllExceptLast[nIndexBits - 1];
        int markTargets[] = {nIndexBits - 1};
        for (int i = 0; i < nIndexBits - 1; ++i)
        {
            controlsAll[i] = i;
            controlsAllExceptLast[i] = i;
        }
        controlsAll[nIndexBits - 1] = nIndexBits - 1;

        // custatevec handle initialization
        custatevecSamplerDescriptor_t sampler;
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // initialize the state vector
        CHECK_CUSTATEVECTOR(custatevecInitializeStateVector(
            handle, d_sv, cuStateVecCudaDataType, nIndexBits, CUSTATEVEC_STATE_VECTOR_TYPE_ZERO));

        // H to all qubits
        for (int i = 0; i < nIndexBits; ++i)
        {
            int targets[] = {i};
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        // H to all qubits

        for (int i = 0; i < 10; ++i)
        {
            // mark
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, zMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast, nullptr,
                nIndexBits - 1, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            // Diffusion
            // H->all, X->all, cz->allexceptLast mark, x->all, H->all
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, xMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, zMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast, nullptr,
                nIndexBits - 1, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, xMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
        }

        // create sampler and check the size of external workspace
        CHECK_CUSTATEVECTOR(custatevecSamplerCreate(
            handle, d_sv, cuStateVecCudaDataType, nIndexBits, &sampler, nMaxShots,
            &extraWorkspaceSizeInBytes));

        // allocate external workspace if necessary
        if (extraWorkspaceSizeInBytes > 0)
            CHECK_CUDA(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

        // sample preprocess
        CHECK_CUSTATEVECTOR(custatevecSamplerPreprocess(
            handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes));

        // sample bit strings
        CHECK_CUSTATEVECTOR(custatevecSamplerSample(
            handle, sampler, bitStrings, bitOrdering, bitStringLen, randnums, nShots,
            CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

        // destroy descriptor and handle
        CHECK_CUSTATEVECTOR(custatevecSamplerDestroy(sampler));

        //  destroy handle
        CHECK_CUSTATEVECTOR(custatevecDestroy(handle));
        if (extraWorkspaceSizeInBytes)
            CHECK_CUDA(hipFree(extraWorkspace));

        auto stop_m = std::chrono::high_resolution_clock::now();
        auto duration_m = std::chrono::duration<double>(stop_m - start_m);
        std::cout << "Time = " << duration_m.count() << std::endl;
    }

    CHECK_CUDA(hipMemcpy(h_sv, &d_sv[svSize - 1], 1 * sizeof(cuType),
                          hipMemcpyDeviceToHost));
    //----------------------------------------------------------------------------------------------

    std::cout << std::fixed << std::setprecision(15) << "(" << h_sv[0].x << "," << h_sv[0].y << ")" << "\n";
    // for (int i = 0; i < svSize; ++i)
    // {
    //     std::cout << std::fixed << std::setprecision(15) << "(" << d_sv[i].x << "," << d_sv[i].y << ")" << "\n";
    // }
    std::cout << "\n\n\n";

    // printDeviceArray(d_sv, svSize);
    CHECK_CUDA(hipFree(d_sv));

    return EXIT_SUCCESS;
}