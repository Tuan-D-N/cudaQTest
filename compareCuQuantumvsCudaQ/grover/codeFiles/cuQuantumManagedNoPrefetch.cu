#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <string>
#include <custatevec.h> // custatevecInitializeStateVector
#include <hip/hip_complex.h>
#include <random>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iomanip>

#define INV_SQRT2 (0.7071067811865475) // Approximation of 1/sqrt(2)

// Macro to check CUDA API errors
#define CHECK_CUDA(func)                                                          \
    {                                                                             \
        hipError_t status = (func);                                              \
        if (status != hipSuccess)                                                \
        {                                                                         \
            printf("CUDA API failed at line %d in file %s with error: %s (%d)\n", \
                   __LINE__, __FILE__, hipGetErrorString(status), status);       \
            return EXIT_FAILURE;                                                  \
        }                                                                         \
    }

// Macro to check cuSPARSE API errors
#define CHECK_CUSPARSE(func)                                                     \
    {                                                                            \
        hipsparseStatus_t status = (func);                                        \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                   \
        {                                                                        \
            printf("CUSPARSE API failed at line %d in file %s with error: %d\n", \
                   __LINE__, __FILE__, status);                                  \
            return EXIT_FAILURE;                                                 \
        }                                                                        \
    }

// Macro to check cuSPARSE API errors
#define CHECK_CUSTATEVECTOR(func)                                                     \
    {                                                                                 \
        custatevecStatus_t status = (func);                                           \
        if (status != CUSTATEVEC_STATUS_SUCCESS)                                      \
        {                                                                             \
            printf("CUSTATEVECTOR API failed at line %d in file %s with error: %d\n", \
                   __LINE__, __FILE__, status);                                       \
            return EXIT_FAILURE;                                                      \
        }                                                                             \
    }

#define CHECK_BROAD_ERROR(integer)                                    \
    {                                                                 \
        if (integer != 0)                                             \
        {                                                             \
            printf("Broad CUDA ERROR failed at line %d in file %s\n", \
                   __LINE__, __FILE__);                               \
            return EXIT_FAILURE;                                      \
        }                                                             \
    }

void generateRandomArray(double *arr, std::size_t size)
{
    std::random_device rd;                                   // Seed
    std::mt19937 gen(rd());                                  // Mersenne Twister RNG
    std::uniform_real_distribution<double> dist(0.0f, 1.0f); // Range [0, 1)

    for (std::size_t i = 0; i < size; ++i)
    {
        arr[i] = dist(gen);
    }
}

int main(int argc, char const *argv[])
{
    const int nIndexBits = 1 < argc ? atoi(argv[1]) : 30;
    // const auto cuStateVecComputeType = CUSTATEVEC_COMPUTE_64F;
    // auto cuStateVecCudaDataType = HIP_C_64F;
    // using cuType = hipDoubleComplex;
    using cuType = hipComplex;
    const auto cuStateVecComputeType = CUSTATEVEC_COMPUTE_32F;
    const auto cuStateVecCudaDataType = HIP_C_32F;

    const int svSize = (1 << nIndexBits);

    const int nShots = 100;
    const int nMaxShots = nShots;
    std::vector<int> bitOrdering = std::vector<int>(nIndexBits, 0);
    for (int i = 0; i < nIndexBits; ++i)
    {
        bitOrdering[i] = i;
    }
    const int bitStringLen = nIndexBits;
    custatevecIndex_t bitStrings[nShots];
    double randnums[nShots] = {};
    generateRandomArray(randnums, nShots);

    cuType xMat[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};
    cuType zMat[] = {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {-1.0, 0.0}};
    cuType hMat[] = {{INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {-INV_SQRT2, 0.0}};

    cuType *d_sv;
    CHECK_CUDA(hipMallocManaged((void **)&d_sv, svSize * sizeof(cuType)));

    //----------------------------------------------------------------------------------------------

    {
        auto start_m = std::chrono::high_resolution_clock::now();

        // std::vector<int> controlsAll = std::vector<int>(nIndexBits, 0);
        std::vector<int> controlsAllExceptLast = std::vector<int>(nIndexBits - 1, 0);
        int markTargets[] = {nIndexBits - 1};
        for (int i = 0; i < nIndexBits - 1; ++i)
        {
            // controlsAll[i] = i;
            controlsAllExceptLast[i] = i;
        }
        // controlsAll[nIndexBits - 1] = nIndexBits - 1;

        // custatevec handle initialization
        custatevecSamplerDescriptor_t sampler;
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Init to zero state
        hipMemPrefetchAsync(d_sv, svSize * sizeof(cuType), hipCpuDeviceId);
        d_sv[0] = {1, 0};
        for (int i = 1; i < svSize; ++i)
        {
            d_sv[i] = {0, 0};
        }

        // Prefetch memory to the GPU
        int device;
        CHECK_CUDA(hipGetDevice(&device)); // Get the current GPU device
        std::cout << "Current GPU: " << device << "\n";
        hipMemPrefetchAsync(d_sv, svSize * sizeof(cuType), device);

        // H to all qubits
        for (int i = 0; i < nIndexBits; ++i)
        {
            int targets[] = {i};
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        // H to all qubits

        for (int i = 0; i < 10; ++i)
        {
            // mark
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, zMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast.data(), nullptr,
                nIndexBits - 1, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            // Diffusion
            // H->all, X->all, cz->allexceptLast mark, x->all, H->all
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, xMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, zMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast.data(), nullptr,
                nIndexBits - 1, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, xMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
        }

        // create sampler and check the size of external workspace
        CHECK_CUSTATEVECTOR(custatevecSamplerCreate(
            handle, d_sv, cuStateVecCudaDataType, nIndexBits, &sampler, nMaxShots,
            &extraWorkspaceSizeInBytes));

        // allocate external workspace if necessary
        if (extraWorkspaceSizeInBytes > 0)
            CHECK_CUDA(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

        // sample preprocess
        CHECK_CUSTATEVECTOR(custatevecSamplerPreprocess(
            handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes));

        // sample bit strings
        CHECK_CUSTATEVECTOR(custatevecSamplerSample(
            handle, sampler, bitStrings, bitOrdering.data(), bitStringLen, randnums, nShots,
            CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

        // destroy descriptor and handle
        CHECK_CUSTATEVECTOR(custatevecSamplerDestroy(sampler));

        //  destroy handle
        CHECK_CUSTATEVECTOR(custatevecDestroy(handle));
        if (extraWorkspaceSizeInBytes)
            CHECK_CUDA(hipFree(extraWorkspace));

        auto stop_m = std::chrono::high_resolution_clock::now();
        auto duration_m = std::chrono::duration<double>(stop_m - start_m);
        std::cout << "Time = " << duration_m.count() << std::endl;
    }
    //----------------------------------------------------------------------------------------------
    hipMemPrefetchAsync(d_sv, svSize * sizeof(cuType), hipCpuDeviceId);
    for (int i = svSize - 1; i < svSize; ++i)
    {
        // std::cout << std::fixed << std::setprecision(15) << "(" << d_sv[i].x << "," << d_sv[i].y << ")" << "\n";
    }
    // std::cout << "\n\n\n";

    // printDeviceArray(d_sv, svSize);
    CHECK_CUDA(hipFree(d_sv));

    return EXIT_SUCCESS;
}